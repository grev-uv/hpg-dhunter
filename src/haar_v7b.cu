#include "hip/hip_runtime.h"
/*
*  haar_vX is the well synchronized version of haar transform into de GPU.
*  This version work with all the samples as a matrix into de GPU
*  with dimension SAMPLES x (sample_num + data_adjust) (rows x cols)
*  Copyright (C) 2018 Lisardo Fernández Cordeiro <lisardo.fernandez@uv.es>
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2, or (at your option)
*  any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*
*  You should have received a copy of the GNU General Public License
*  along with this program; if not, write to the Free Software
*  Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA.
*
*/
 
/** \file
*  \brief Archivo para procesamiento de diferentes muestras metiladas de ADN.
*
*  Este archivo contiene la definición de las funciones para:
*         ..carga de datos en GPU
*         ..lanzamiento de proceso de transformación en GPU
*         ..kernel en GPU para control de transformación en niveles definidos
*         ..kernel de transformación wavelet del vector seleccionado
*         ..kernel para copiar coeficientes desde vector auxiliar de sincronización
*/

#include <stdio.h>
#include <GL/gl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "data_pack.h"

#define BLOCK_SIZE  1024		// número de hilos por bloque de GPU
#define AJUSTE_PLOT 1//.70        // ajusta eje Y de gráfica a AJUSTE_PLOT
#define DESPLAZAMIENTO_DIBUJO 0.97
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // para gestión de errores en GPU


#define DESPLAZAMIENTO_DIBUJO_X 1           //0.97
#define DESPLAZAMIENTO_DIBUJO_Y 0//0.97        //1
#define WAVELET_COEF            4// coeficientes para wavelet bior-3.1


/** ***********************************************************************************************
  * \fn void gpuAssert(hipError_t, char*, int, bool)
  *  \brief Función responsable de recoger error en GPU y mostrarlo
  *  \param code	código de error de la GPU
  *  \param *file	fichero donde se produce el error
  *  \param line	línea de código donde se produce el error
  *  \param abort	indica si se sale del programa
  * ***********************************************************************************************
  */
extern "C"
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
        fprintf(stderr, "GPUassert: %s %s %d\n\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

/** ***********************************************************************************************
  * \fn void copyValuesTotal(float*, float *, int, int)
  *  \brief función "hija" en GPU responsable de la copia de los datos del segmento a transformar
  *         proporcionando sincronización a nivel GRID
  *  \param *haar	puntero a vector de datos original
  *  \param *aux	puntero a vector de datos a transformar
  *  \param num		numero datos totales a transformar
  *  \param pi  	posicion inicial de copia - offset
  * ***********************************************************************************************
  */
extern "C"
__global__
void copyValuesTotal(float *haar, float *aux, int num, int posicion_inicial)
{
	// variables ------------------------------------------------------------------------------
	int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector

	// copiar todos los valores de haar en aux
	if (index < num)
        	aux[index] = haar[index + posicion_inicial];
}


/** ***********************************************************************************************
  * \fn void copyValues(float*, float *, int)
  *  \brief función "hija" en GPU responsable de la copia de los valores escalados
  *         proporcionando sincronización a nivel GRID
  *  \param *aux	puntero a vector de datos a transformar
  *  \param *temp	puntero a vector de datos temporales a copiar
  *  \param num     numero datos a copiar
  * ***********************************************************************************************
  */
extern "C"
__global__
void copyValues(float *aux, float *temp, int num)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector

    // copiar todos los valores de haar en aux
    if (index < num)
        aux[index] = temp[index];
}


/** ***********************************************************************************************
  * \fn void transform(float*, int, int, int)
  *  \brief función "hija" en GPU responsable de la transformación wavelet de un vector
  *         proporcionando sincronización a nivel GRID.
  *  \param *aux	puntero a vector de datos a transformar
  *  \param *temp	puntero a vector de resultados intermedios
  *  \param num		número de posiciones del vector
  * ***********************************************************************************************
  */
extern "C"
__global__
void transform(float *aux, float *temp, int num)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector
/*    float f   = 0.7071067811865476;                     // coeficiente haar wavelet
    float aux1;                                         // variables auxiliares de sincronización
    int idx;                                            // indice auxiliar para guardar dato

    // transformada haar en paralelo sobre el vector recibido -------------------------------------
    if (index < num)
    {
        if ((index & 0x01) == 0)	// solo los hilos con índice par (0, 2, 4, ...)
        {
            idx = index * 0.5;

            aux1 = (aux[index] + aux[index + 1]) * f;	// escalado (filtro paso-bajo)

            temp[idx]  = aux1;
        }
    }
*/


    float f[WAVELET_COEF];
    float bior31[4]  = {-0.3535533905932738, 1.0606601717798214, 1.0606601717798214, -0.3535533905932738};
    float spline[5]  = {-0.1767766953, 0.3535533906, 0.7071067812, 0.3535533906, -0.1767766953};
    float bior33[8]  = {0.06629126073623884, -0.19887378220871652, -0.15467960838455727, 0.9943689110435825,
                        0.9943689110435825, -0.15467960838455727, -0.19887378220871652, 0.06629126073623884};
    float bior35[12] = {-0.013810679320049757, 0.04143203796014927, 0.052480581416189075, -0.26792717880896527,
                       -0.07181553246425874, 0.966747552403483, 0.966747552403483, -0.07181553246425874,
                       -0.26792717880896527, 0.052480581416189075, 0.04143203796014927, -0.01381067932004975};

    switch (WAVELET_COEF)
    {
    case 4:
        for (int i = 0; i < WAVELET_COEF; i++)
            f [i] = bior31 [i];
        break;
    case 5:
        for (int i = 0; i < WAVELET_COEF; i++)
            f [i] = spline [i];
        break;
    case 8:
        for (int i = 0; i < WAVELET_COEF; i++)
            f [i] = bior33 [i];
        break;
    case 12:
        for (int i = 0; i < WAVELET_COEF; i++)
            f [i] = bior35 [i];
        break;
    default:
        ;
    }

    float aux1 = 0.0;                                      // variable auxiliar de almacenamiento intermedio de resultado
    int idx;                                               // indice auxiliar para guardar dato

    if (index < num + (WAVELET_COEF - 2))
    {
        if ((index & 0x01) == 0)
        {
            idx = floorf(index * 0.5);

            for (int i = -(WAVELET_COEF - 2); i < 2; i++)
            {
                if (index + i < 0)
                    aux1 += 0.0;
                else if (index + i > num)
                    aux1 += 0.0;
                else
                    aux1 += aux[index + i] * f[i + (WAVELET_COEF - 2)];
            }

            temp[idx] = aux1;
        }
    }

}

/** ***********************************************************************************************
  * \fn void array2Plot(float*, float*, int)
  *  \brief función "hija" en GPU responsable de crear el array para ploteado.
  *  \param *aux	puntero a vector de datos tranformados
  *  \param *glPtr	puntero a vector de datos para plotear
  *  \param num		número de datos a plotear
  *  \param max     valor máximo de cálculo para escalar todos los valores entre 0 y 2
  *  \param hilo    hilo que gestiona el ploteado
  * ***********************************************************************************************
  */
extern "C"
__global__
void array2Plot(float *aux, float *glPtr, int num, float *max, int hilo)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector
    int idx;

    // copia todos los valores de aux en glPtr duplicando las posiciones anexas para crear una
    // gráfica en escalón propia de la transformada wavelet haar
    if (index < num)
    {
        idx = hilo * num + index;

        glPtr[idx * 4]     = (index * 2.0 / num) - DESPLAZAMIENTO_DIBUJO_X;     // eje x de -1 a 1
        glPtr[idx * 4 + 1] = aux[index] / max[0] * AJUSTE_PLOT - DESPLAZAMIENTO_DIBUJO_Y;              // eje y de 0 a 2
        glPtr[idx * 4 + 2] = ((index+1) * 2.0 / num) - DESPLAZAMIENTO_DIBUJO_X; // eje x de -1 a 1
        glPtr[idx * 4 + 3] = aux[index] / max[0] * AJUSTE_PLOT - DESPLAZAMIENTO_DIBUJO_Y;              // eje y de 0 a 2

    }
}


/** ***********************************************************************************************
  * \fn void maxVal(float*, float*, int, int)
  *  \brief función "hija" en GPU responsable de encontrar el máximo valor de cálculo
  *                 por método de reducción con memoria local a nivel de hilo, encontrando
  *                 el máximo valor por bloque.
  *  \param *aux_c	puntero a vector de datos
  *  \param *max	puntero a vector de maximos encontrados por bloque
  *  \param num		número de datos
  * ***********************************************************************************************
  */
extern "C"
__global__
void maxVal(float *aux_c, float *max, int num)
{
    extern __shared__ float sdata[];

    // para aprovechar el total de los hilos en la primera operación de búsqueda
    // se direcciona al doble de la capacidad de un bloque (blockDim.x * 2)
    unsigned int index = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int tid = threadIdx.x;

    if (index < num)
    {
        // la primera carga de datos a la memoria local, se realiza buscando el máximo
        // de cada parte de los datos direccionados
        if (aux_c[index] >= aux_c[index + blockDim.x])
            sdata[tid] = aux_c[index];
        else
            sdata[tid] = aux_c[index + blockDim.x];

        __syncthreads();

        // a partir de la carga condicionada de los datos en memoria locas,
        // se busca el máximo del bloque por reducción
        for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1)
        {
            if (tid < i) // && (tid + i + (blockDim.x * blockIdx.x)) < index) // / 2)
                if (sdata[tid] < sdata[tid + i])
                    sdata[tid] = sdata[tid + i];

            __syncthreads();
        }

        // guarda el resultado de cada bloque para el siguiente paso
        if (tid == 0)
            max[blockIdx.x] = sdata[0];
    }
}

/** ***********************************************************************************************
  * \fn void maxGlobal(float*, int, int)
  *  \brief función "hija" en GPU responsable de encontrar el máximo valor de cálculo
  *                 por método de reducción con memoria local a nivel de hilo, encontrando
  *                 el máximo valor entre los máximos encontrados de cada muestra.
  *  \param *temp	puntero a vector de máximos
  *  \param pitch	número de bytes por muestra reservados para el vector temporal
  *  \param samples número de muestras
  * ***********************************************************************************************
  */
extern "C"
__global__
void maxGlobal(float *temp, size_t pitch, int samples)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x;

    // solo un hilo se encarga de buscar el máximo entre los máximos encontrados para cada muestra
    if (index == 0)
    {
        for (int i = 1; i < samples; i++)
        {
            if (temp[0] < temp[i * pitch / sizeof(float)])
                temp[0] = temp[i * pitch / sizeof(float)];
        }
    }
}


/** ***********************************************************************************************
  * \fn void wavedec(float*, float**, int, int, int, int, int)
  *  \brief Función principal en GPU responsable de calcular y ordenar las partes del vector
  *         para su transformación wavelet multinivel.
  *  \param *haar	puntero a matriz de datos a transformar
  *  \param *aux	puntero a matriz de coeficiente auxiliares para ayuda a la sincronización
  *  \param *temp	puntero a matriz de cálculos temporales de ayuda a la sincronización
  *  \param pitch	desplazamiento óptimo en memoria GPU para alojar cada muestra	
  *  \param pitch_2	desplazamiento óptimo en memoria GPU para alojar cálculo auxiliar
  *  \param pitch_3	desplazamiento óptimo en memoria GPU para alojar cálculo temporal
  *  \param n		número total de posiciones del vector
  *  \param l		número de niveles a computar
  *  \param samples número de muestras a analizar
  *  \param pi      posición inicial del segmento de datos a analizar
  *  \param *glPtr  puntero a array de datos para dibujar con openGL
  * ***********************************************************************************************
  */
extern "C"
__global__ 
void wavedec(float *haar, float *aux, float *temp,
             size_t pitch, size_t pitch_2, size_t pitch_3,
             int n, int l, int samples, int pi, float *glPtr)
{
    // variables ----------------------------------------------------------------------------------
	int index_X = threadIdx.x + blockIdx.x * blockDim.x;	// indice de hilos sobre todo el vector
    int level   = 0;                                        // número de nivel
    int num     = n;                                        // número de posiciones en vector
    int hilo;                                               // guarda el hilo asignado para que se resposabilice de todo el proceso
    int nume    = 0;                                        // numero de posiciones antes de ajuste para cálculo de nuevo nivel
                                                            //     evita que los datos de ploting hagan rayas por descuadre en posición


    // limita el número de hilos al de muestras ---------------------------------------------------
	if (index_X < samples)
	{
		hilo = index_X;		// cada hilo se responsabiliza de una misma muestra

		if (hilo == index_X)		
		{
            // separar los datos por muestras - - - - - - - - - - - - - - - - - - - - - - - - - - -
			float *haar_c = (float *)((char *)haar + index_X * pitch);
            float *aux_c  = (float *)((char *)aux  + index_X * pitch_2);
            float *temp_c = (float *)((char *)temp + index_X * pitch_3);

			__syncthreads(); 

            // llamada a función hija para copiar segmento de vector a transformar
            copyValuesTotal<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(haar_c,
                                                                               aux_c,
                                                                               num,
                                                                               pi);


            // procesamiento multinivel del vector de datos ---------------------------------------
			// repite la transformación tantas veces como niveles se han solicitado
            while (level < l && num >= 2)
            {
				// llamada a función hija para transformación del nivel correspondiente
				// con esta división en padre-hijo, se consigue sincronizar cada nivel 
				// \param	<<<((datos_x_muestra + num_hilos_bloque-1) / num_hilos_bloque),
				// 		numero hilos por bloque>>>
                transform<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(aux_c,
                                                                             temp_c,
                                                                             num);


                // actualizar variables de nivel  - - - - - - - - - - - - - - - - - - - - - - - - -
                level += 1;
                num    = ceilf(num * 0.5);


                // llamada a función hija para copiar resultados en vector auxiliar
                copyValues<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(aux_c,
                                                                              temp_c,
                                                                              num);


                // actualiza el número de datos para el siguiente nivel - - - - - - - - - - - - - -
                nume = num;
                if ((num & 01) == 1)
                {
                    num++;
                    aux_c[num] = 0;
                }
            }



            // hallar el valor máximo de todas las muestras para ajustar ploteado, para ello:
            // busca el máximo donde están los datos calculados (aux_c) y aprovecha para pasarlos a temp_c
            maxVal<<<(nume + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(aux_c,
                                                                                                   temp_c,
                                                                                                   nume);

            // una vez los datos en temp_c, si el número de datos es mayor que la capcidad de un bloque,
            // repite la operación anterior pero dentro de temp_c, dividiendo el tramo en dos
            float maximos = ceilf(nume * 1.0 / BLOCK_SIZE);
            while (maximos > 1.0)
            {
                // halla el máximo por parejas almacenando el resultado en la mitad superior
                maxVal<<<(maximos + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(temp_c,
                                                                                                          &temp_c[pitch_3 / (sizeof(float) * 2)],
                                                                                                          ceilf(maximos));
                __syncthreads();

                maximos = ceilf(maximos / BLOCK_SIZE);

                // copia los datos en la mitad inferior para volver a reducirlos
                copyValues<<<(maximos + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(temp_c,
                                                                                  &temp_c[pitch_3 / (sizeof(float) * 2)],
                                                                                  ceilf(maximos));

            }

            __syncthreads();

            // una vez obtenido el máximo por cada muestra, se busca el máximo de todos los valores
            if (hilo == 0)
                maxGlobal<<< 1, 1 >>>(temp,
                                      pitch_3,
                                      samples);

            __syncthreads();

            // rellenar el array con datos para dibujar desde openGL
            // llamada a función hija para rellenar datos de gráfica
            array2Plot<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(aux_c,
                                                                          glPtr,
                                                                          nume,
                                                                          temp,
                                                                          hilo);
        }
	}
}

/** ***********************************************************************************************
  * \fn void cuda_send_data(datos_cuda &)
  *  \brief Función para enviar los datos a la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_send_data(datos_cuda &cuda_data)
{
    // reserva espacio en GPU para el vector a transformar y copia matriz de datos ----------------
    // devuelve valor de desplazamiento (pitch) óptimo para gestión de memoria adecuada
    // en función de la cantdad de datos a alojar
    // \param 	puntero a posición memoria GPU,
    //          desplazamiento óptimo devuelto por CUDA,
    //          cantidad de bytes a reservar por fila,
    //          número de muestras (filas)
    gpuErrchk(hipMallocPitch(&cuda_data.d_haar,
                              &cuda_data.pitch,
                              (cuda_data.sample_num + cuda_data.data_adjust) * sizeof(float),
                              cuda_data.samples));


    gpuErrchk(hipMallocPitch(&cuda_data.d_aux,
                              &cuda_data.pitch_2,
                              (cuda_data.sample_num + cuda_data.data_adjust) * sizeof(float),
                              cuda_data.samples));


    // envío de datos a GPU -----------------------------------------------------------------------
    // \param	puntero a posición de memoria GPU,
    //          desplazamiento óptimo,
    //          puntero a posición de datos en CPU a enviar a GPU,
    //          cantidad de bytes a enviar por muestra,
    //          cantidad de bytes a alojar por muestra,
    //          número de filas (muestras)
    gpuErrchk(hipMemcpy2D( cuda_data.d_haar,
                            cuda_data.pitch,
                            cuda_data.mc_full[0],
                            cuda_data.sample_num * sizeof(float),
                            cuda_data.sample_num * sizeof(float),
                            cuda_data.samples,
                            hipMemcpyHostToDevice));

}


/** ***********************************************************************************************
  * \fn void cuda_main(datos_cuda &)
  *  \brief Función para procesar los datos en la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_main(datos_cuda &cuda_data)
{
    // reserva TODA la memoria CONTIGUA para la matriz de muestras tranformadas -------------------
    // para trasvase de datos entre GPU y CPU con CUDA, la matriz debe ser contigua completa
    cuda_data.h_haar_C = new float*[cuda_data.samples];                             // reservar punteros a filas
    cuda_data.h_haar_C[0] = new float[cuda_data.samples * (cuda_data.h_haar_L[0])];	// reservar toodos los datos (rows * cols)
    for (int i = 1; i < cuda_data.samples; i++)                                     // asignar valor a punteros de fila
        cuda_data.h_haar_C[i] = cuda_data.h_haar_C[i - 1] + cuda_data.h_haar_L[0];


    // reserva memoria para cálculos temporales en GPU --------------------------------------------
    float *d_temp;
    size_t pitch;
    gpuErrchk(hipMallocPitch(&d_temp,
                              &pitch,
                              (cuda_data.sample_num + 1) * sizeof(float) * 0.7,
                              cuda_data.samples));


    // transforma el número de muestras elegida ---------------------------------------------------
	// realiza la transformación en la GPU del conjunto de muestras cargado
	// \param	<<< número de bloques a utilizar,
    //          número de hilos por bloque >>> (máximo 1024 para PASCAL GTX 1080)
	// \param	puntero a datos a transformar alojados en GPU,
    //          desplazamiento óptimo de datos por fila,
    //          número de datos por muestra (fila) a transformar,
    //          ajuste de longitud de muestra por número impar al dividir la muestra
    wavedec<<<1, cuda_data.samples>>>(cuda_data.d_haar,
                                      cuda_data.d_aux,
                                      d_temp,
                                      cuda_data.pitch,
                                      cuda_data.pitch_2,
                                      pitch,
                                      cuda_data.sample_num,
                                      cuda_data.levels,
                                      cuda_data.samples,
                                      cuda_data.rango_inferior,
                                      (float *)cuda_data.d_glPtr);

    // espera a que la GPU termine el trabajo - - - - - - - - - - - - - - - - - - - - - - - - - - -
    gpuErrchk(hipDeviceSynchronize());


    // recupera el resultado de la transformación en memoria GPU a memoria CPU- - - - - - - - - - -
	// \param	puntero a matriz de datos a guardar en CPU,
    //          cantidad de bytes a guardar por muestra,
    //          puntero a datos para copiar de GPU,
    //          desplazamiento óptimo de datos por fila en GPU,
    //          cantidad de bytes en GPU a copiar por muestra,
    //          número de muestras (filas)
    gpuErrchk(hipMemcpy2D(	cuda_data.h_haar_C[0],
                            cuda_data.h_haar_L[0] * sizeof(float),
                            cuda_data.d_aux,
                            cuda_data.pitch,
                            cuda_data.h_haar_L[0] * sizeof(float),
                            cuda_data.samples,
                            hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy( cuda_data.d_max,
                          d_temp,
                          cuda_data.samples * sizeof(float),
                          hipMemcpyDeviceToHost));


    //libera la memoria temporal utilizada para cálculos intemedios
    hipFree(d_temp);
}

/** ***********************************************************************************************
  * \fn void *cuda_init()
  *  \brief Función para inicializar la gpu
  * ***********************************************************************************************
  */
void cuda_init()
{
    int deviceCount = 0;
    int cudaDevice  = 0;
    char cudaDeviceName [100];
    hipDeviceProp_t prop;
    hipInit(0);
    hipGetDeviceCount(&deviceCount);
    hipDeviceGet(&cudaDevice, 0);
    hipDeviceGetName(cudaDeviceName, 100, cudaDevice);
    hipGetDeviceProperties(&prop, cudaDevice);

    if (hipChooseDevice(&cudaDevice, &prop) != hipSuccess)
        puts("failed to choose device");
    if (cudaGLSetGLDevice(cudaDevice) != hipSuccess)
        puts("failed to set gl device");

    printf("Number of devices: %u \t cuda device: %d\n", deviceCount, cudaDevice);
    printf("Device name: %s\n", cudaDeviceName);
    printf("Warp size: %u\n", prop.warpSize);
}

/** ***********************************************************************************************
  * \fn void cuda_end(data buf)
  *  \brief Función para liberar memoria de la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_end(datos_cuda &cuda_data)
{
    //libera la memoria de la gpu utilizada para cálculos intemedios
    hipFree(cuda_data.d_haar);
    hipFree(cuda_data.d_aux);
}

/** ***********************************************************************************************
  * \fn void *cuda_registerBuffer(GLuint buf)
  *  \brief Función para registrar el vínculo de cuda con opengl
  *  \param buf buffer donde se alojan los datos para procesar
  * ***********************************************************************************************
  */
void *cuda_registerBuffer(GLuint buf)
{
    hipGraphicsResource *res = 0;

    if (hipGraphicsGLRegisterBuffer(&res, buf, hipGraphicsRegisterFlagsNone) != hipSuccess)
        printf("Fallo en el registro del buffer %u\n", buf);

    return res;
}

/** ***********************************************************************************************
  * \fn void cuda_unregisterBuffer(void *res)
  *  \brief Función para desvincular opengl de cuda
  *  \param *res    referencia al vínculo
  * ***********************************************************************************************
  */
void cuda_unregisterBuffer(void *res)
{
    if (hipGraphicsUnregisterResource((hipGraphicsResource *) res) != hipSuccess)
        puts("Fsllo eliminando el registro de recursos para el buffer");
}

/** ***********************************************************************************************
  * \fn void *cuda_map(void *res)
  *  \brief Función para mapear los datos cuda sobre opengl
  *  \param *res    variable de vínculo con los datos cuda
  * ***********************************************************************************************
  */
void *cuda_map(void *res)
{
    if (hipGraphicsMapResources(1, (hipGraphicsResource **) &res) != hipSuccess, 0)
    {
        puts("Fallo en el mapeado de recursos");
        return 0;
    }

    void *devPtr = NULL;

    size_t size;

    if (hipGraphicsResourceGetMappedPointer(&devPtr, &size, (hipGraphicsResource *) res) != hipSuccess)
    {
        puts("Fallo en la adquisición del puntero del dispositivo ");
        return 0;
    }

    return devPtr;
}

/** ***********************************************************************************************
  * \fn void cuda_unmap(void *res)
  *  \brief Función para liberar el mapeo de datos
  *  \param *res    variable de vínculo con los datos cuda
  * ***********************************************************************************************
  */
void cuda_unmap(void *res)
{
    if (hipGraphicsUnmapResources(1,(hipGraphicsResource **) &res, 0) != hipSuccess)
        puts("Fallo en el desmapeado de recursos");
}

/** ***********************************************************************************************
  * \fn void calculo_haar_L(datos_cuda &cuda_data)
  *  \brief Función para calcular el número de datos en el nivel dado y el ajuste por impares
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_calculo_haar_L(datos_cuda &cuda_data)
{
    // cálculo de número de coeficientes por nivel y del ajuste de paso entre escala y coeficiente
    cuda_data.h_haar_L.push_front(cuda_data.sample_num);	// última posición guarda el total de posiciones por muestra

    // para cada nivel se divide por dos la cantidad de posiciones del nivel anterior -------------
    // redondeando al alza y actualizando el ajuste cuando sea impar
    for (int fila = cuda_data.levels; fila > 0; fila--)
    {
        if (ceil(cuda_data.h_haar_L.front() * 0.5 >= 2))
        {
            cuda_data.h_haar_L.push_front(ceil(cuda_data.h_haar_L.front() * 0.5));
            if (fila > 0 && cuda_data.h_haar_L[1] != cuda_data.sample_num)
                cuda_data.data_adjust += size_t(2 * cuda_data.h_haar_L.front() - cuda_data.h_haar_L[1]);
        }
        else
            break;
    }
    cuda_data.h_haar_L.push_front(cuda_data.h_haar_L.front());	// primera posición coincide con el número de datos de escala
}
